
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979323846

__global__ void gpu_mmc(hiprandState *states, int *global_count, int n) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int local_count = 0;
    
    hiprandState localState = states[idx];
    
    for(int i = idx; i < n; i += stride) {
        float x = hiprand_uniform(&localState);
        float y = hiprand_uniform(&localState);
        if(x*x + y*y <= 1.0f) {
            local_count++;
        }
    }
    
    atomicAdd(global_count, local_count);
    states[idx] = localState;

}

__global__ void setup_kernel(hiprandState *state, int n) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= n) return;
    hiprand_init(clock64(), idx, 0, &state[idx]);

}

int main(int argc, char *argv[]) {
    
    int n_points = atoll(argv[1]);
    int n_blocks = 1024;
    int s_blocks = 128;
    int *d_count, h_count = 0;

    float time;

    hiprandState *d_states;
    hipError_t nb_error;
    hipEvent_t start, stop;   

    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;  
    
    int bytes = n_blocks*s_blocks*sizeof(hiprandState);

    // 
    hipMalloc(&d_states, bytes);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 1: %s\n", hipGetErrorString(nb_error));
    // 
    hipMalloc(&d_count, sizeof(int));
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 2: %s\n", hipGetErrorString(nb_error));
    // 
    hipMemset(d_count, 0, sizeof(int));
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 3: %s\n", hipGetErrorString(nb_error));
    // 

    setup_kernel<<<n_blocks, s_blocks>>>(d_states, n_blocks * s_blocks);
    
    gpu_mmc<<<n_blocks, s_blocks>>>(d_states, d_count, n_points);

    hipDeviceSynchronize();    
        
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

    double pi = 4.0 * (double)h_count / (double)n_points;
    double error = fabs(pi - PI)/PI * 100.0;
    
    printf("pi: %.15f\n", pi);
    printf("erro: %.10f%%\n", error);
    printf("tempo: %3.1fms\n", time);
    
    hipFree(d_states);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 4: %s\n", hipGetErrorString(nb_error));
    
    hipFree(d_count);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 5: %s\n", hipGetErrorString(nb_error));
    
    hipEventDestroy(start);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 6: %s\n", hipGetErrorString(nb_error));
    
    hipEventDestroy(stop);
    nb_error = hipGetLastError();
    if(nb_error != hipSuccess) printf("Error 7: %s\n", hipGetErrorString(nb_error));
    
    return 0;
    
}